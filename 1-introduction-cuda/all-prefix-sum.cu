#include "hip/hip_runtime.h"
// Copyright 2023 Pierre Talbot

#include "./utility.hpp"
#include <stdlib.h>
#include <cstdio>

void scan_sequential(int* arr, int* new_arr, int identity_element, size_t arr_size) {
  new_arr[0] = identity_element;

  for(size_t i = 1; i < arr_size; ++i) {
    new_arr[i] = arr[i - 1] + new_arr[i];
  }

}


__global__ void accu_sum(int* arr, int* arr_acc, size_t size) {
  for(size_t i = 0; i < size; ++i) {
    arr_acc[i] = arr_acc[i] + arr[i];
  }
}

int main(int argc, char** argv) {
  // std::vector<int> vec = {1, 2, 3, 4};

  int* arr = (int *)malloc(64 * sizeof(int));
  int* local_scan;

  CUDIE(hipMallocManaged(&local_scan, sizeof(int) * 64));

  // for(int )

  // accu_sum<<<1, 64>>>();
  hipDeviceSynchronize();
  return 0;
}
